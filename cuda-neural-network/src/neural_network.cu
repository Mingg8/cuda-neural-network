#include "hip/hip_runtime.h"
#include "neural_network.hh"
#include "nn_utils/nn_exception.hh"
#include <math.h>

#include <chrono>

using namespace std;

__global__ void normalization(float* pnts, float* coeff, float* n_pnts, int x, int y) {
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < x && col < y) {
		n_pnts[row * y + col] = pnts[row * y + col] * coeff[col] + coeff[col + 3];
	}
}

__global__ void unnormalization(float* pnts, float* coeff, float* n_pnts, int x, int y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < x * y) {
		n_pnts[index] = (pnts[index] - coeff[4])  / coeff[0];
	}
}

__global__ void normal_unnormalization(float* pnts, float* coeff, float* n_pnts, int x, int y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < x) {
		float sum = 0.0f;
		float a[3];
		for (size_t i = 0; i < 3; i++) {
			a[i] = pnts[index + i * y] * coeff[i];
			sum += a[i] * a[i];
		}
		for (size_t i = 0; i < 3; i++) {
			n_pnts[index + i * y] = a[i] / sqrt(sum);
		}
		
	}
}

NeuralNetwork::NeuralNetwork(float learning_rate) :
	learning_rate(learning_rate)
{ }

NeuralNetwork::~NeuralNetwork() {
	for (auto layer : layers) {
		delete layer;
	}
}

void NeuralNetwork::addLayer(NNLayer* layer) {
	this->layers.push_back(layer);
}

void NeuralNetwork::forward(Matrix Z, Matrix& output, Matrix& normal) {
	Z.copyHostToDevice();
	Z = this->normalize(Z); // 18 us

	Matrix a1 = layers[0]->forward(Z);
	Matrix a2 = layers[1]->forward(a1); // 326
	a2 = layers[2]->forward(a2); 
	Matrix a3 = layers[3]->forward(a2); // 321
	a3 = layers[4]->forward(a3);
	Matrix a4 = layers[5]->forward(a3); // 319
	a4 = layers[6]->forward(a4);
	output = layers[7]->forward(a4); // 18
	output = this->unnormalize(output);

	normal = layers[7]->normal(a4); // N x 1
	normal = layers[6]->normal(normal); // (64 x 1) x (N x 1)'
	normal = layers[5]->normal_relu(a3, normal);
	normal = layers[4]->normal(normal);
	normal = layers[3]->normal_relu(a2, normal);
	normal = layers[2]->normal(normal);
	normal = layers[1]->normal_relu(a1, normal);
	normal = layers[0]->normal(normal);
	normal = unnormalize_normal(normal);
}
std::vector<NNLayer*> NeuralNetwork::getLayers() const {
	return layers;
}


void NeuralNetwork::setCoeffs(Matrix& input, Matrix& output) {
	input_coeff = input;
	output_coeff = output;
}

Matrix NeuralNetwork::normalize(Matrix &pnts) {
	Matrix normalized_pnts;
	normalized_pnts.allocateMemoryIfNotAllocated(pnts.shape);

	dim3 block_size(8, 8);
	dim3 num_of_blocks( (pnts.shape.x + block_size.x - 1) / block_size.x,
						(pnts.shape.y + block_size.y - 1) / block_size.y);
	normalization<<<num_of_blocks, block_size>>>(pnts.data_device.get(),
												input_coeff.data_device.get(),
												normalized_pnts.data_device.get(),
												pnts.shape.x, pnts.shape.y);
	return normalized_pnts;
}

Matrix NeuralNetwork::unnormalize(Matrix &pnts) {
	Matrix normalized_pnts;
	normalized_pnts.allocateMemoryIfNotAllocated(pnts.shape);
	dim3 block_size(256);
	dim3 num_of_blocks((pnts.shape.y * pnts.shape.x + block_size.x - 1) / block_size.x);
	unnormalization<<<num_of_blocks, block_size>>>(pnts.data_device.get(),
												output_coeff.data_device.get(),
												normalized_pnts.data_device.get(),
												pnts.shape.x, pnts.shape.y);
	return normalized_pnts;
}

Matrix NeuralNetwork::unnormalize_normal(Matrix &pnts) {
	Matrix normalized_pnts;
	normalized_pnts.allocateMemoryIfNotAllocated(pnts.shape);
	dim3 block_size(256);
	dim3 num_of_blocks((pnts.shape.y + block_size.x - 1) / block_size.x);
	normal_unnormalization<<<num_of_blocks, block_size>>>(pnts.data_device.get(),
												input_coeff.data_device.get(),
												normalized_pnts.data_device.get(),
												pnts.shape.x, pnts.shape.y);
	return normalized_pnts;
}